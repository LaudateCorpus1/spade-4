#include "hip/hip_runtime.h"
using namespace std;

typedef double Data_t;

__global__ void cuda_distance_kernel(double *a, double *b, double *out) {
	int i = threadIdx.x;
	out[i] = abs(a[i] + b[i]);
}

void cuda_distance(const Data_t* a, const Data_t* b, size_t dim, Data_t* out) {
	// device copies
	Data_t *d_a, *d_b, *d_out;

	double size = dim * sizeof(double);
	hipMalloc ((void **) &d_a, size);
	hipMalloc ((void **) &d_b, size);
	hipMalloc ((void **) &d_out, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	cuda_distance_kernel<<<(dim+32-1)/32,32>>>(d_a, d_b, d_out);

	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
	hipDeviceReset();
		
	return;
}